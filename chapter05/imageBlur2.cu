#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"

#define DIM 1024

// these exist on the GPU side
texture<float,2>  texIn;
texture<float,2>  texOut;

__global__ void GOL_kernel( float *dst, bool dstOut ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float   t, l, c, r, b, tl, tr, bl, br, average;
    if (dstOut) {
      t = tex2D(texIn,x,y-1);//top
      l = tex2D(texIn,x-1,y);//left
      c = tex2D(texIn,x,y);//center
      r = tex2D(texIn,x+1,y);//right
      b = tex2D(texIn,x,y+1);//bottom
      tl = tex2D(texIn,x-1,y-1);//top-left
      tr = tex2D(texIn,x+1,y-1);//top-right
      bl = tex2D(texIn,x-1,y+1);//bottom-left
      br = tex2D(texIn,x+1,y+1);//bottom-right
    }else{
      t = tex2D(texOut,x,y-1);//top
      l = tex2D(texOut,x-1,y);//left
      c = tex2D(texOut,x,y);//center
      r = tex2D(texOut,x+1,y);//right
      b = tex2D(texOut,x,y+1);//bottom
      tl = tex2D(texOut,x-1,y-1);//top-left
      tr = tex2D(texOut,x+1,y-1);//top-right
      bl = tex2D(texOut,x-1,y+1);//bottom-left
      br = tex2D(texOut,x+1,y+1);//bottom-right
    }
    average = (t+l+r+b+tl+tr+bl+br+c)/9;
    //boxblur
    dst[offset] = average;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *output_bitmap;
    float           *dev_inSrc;
    float           *dev_outSrc;
    CPUAnimBitmap  *bitmap;

    hipEvent_t     start, stop;
    float           totalTime;
    float           frames;
};

struct hipComplex {
    float   r;
    float   i;
    hipComplex( float a, float b ) : r(a), i(b)  {}
    float magnitude2( void ) { return r * r + i * i; }
    hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};

int julia( int x, int y ) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

void draw( float *ptr ){
    for (int y=0; y<DIM; y++) {
        for (int x=0; x<DIM; x++) {
            int offset = x + y * DIM;

            int juliaValue = julia( x, y );
            ptr[offset*4 + 0] = 255 * juliaValue;
            ptr[offset*4 + 1] = 0;
            ptr[offset*4 + 2] = 0;
            ptr[offset*4 + 3] = 255;
        }
    }
 }

void anim_gpu( DataBlock *d, int ticks ) {
    HANDLE_ERROR( hipEventRecord( d->start, 0 ) );
    dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);
    CPUAnimBitmap  *bitmap = d->bitmap;

    // since tex is global and bound, we have to use a flag to
    // select which is in/out per iteration
    // we maintain this so that cylce speed can be controlled by timesteps or FPS
    volatile bool dstOut = true;
    for (int i=0; i<2; i++) {
        float *out;
        if (dstOut) {
            out = d->dev_outSrc;
        } else {
            out = d->dev_inSrc;
        }
        GOL_kernel<<<blocks,threads>>>( out, dstOut );
        dstOut = !dstOut;
    }
    float_to_color<<<blocks,threads>>>( d->output_bitmap,
                                        d->dev_inSrc );

    HANDLE_ERROR( hipMemcpy( bitmap->get_ptr(),
                              d->output_bitmap,
                              bitmap->image_size(),
                              hipMemcpyDeviceToHost ) );

    HANDLE_ERROR( hipEventRecord( d->stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( d->stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        d->start, d->stop ) );
    d->totalTime += elapsedTime;
    ++d->frames;
    printf( "Average Time per frame:  %3.1f ms\n",
            d->totalTime/d->frames  );
}

// clean up memory allocated on the GPU
void anim_exit( DataBlock *d ) {
    hipUnbindTexture( texIn );
    hipUnbindTexture( texOut );

    HANDLE_ERROR( hipFree( d->dev_inSrc ) );
    HANDLE_ERROR( hipFree( d->dev_outSrc ) );

    HANDLE_ERROR( hipEventDestroy( d->start ) );
    HANDLE_ERROR( hipEventDestroy( d->stop ) );
}


int main( void ) {
    DataBlock   data;
    CPUAnimBitmap bitmap( DIM, DIM, &data );
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    HANDLE_ERROR( hipEventCreate( &data.start ) );
    HANDLE_ERROR( hipEventCreate( &data.stop ) );

    int imageSize = bitmap.image_size();

    HANDLE_ERROR( hipMalloc( (void**)&data.output_bitmap,
                               imageSize ) );

    // assume float == 4 chars in size (ie rgba)
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_inSrc,
                              imageSize ) );
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_outSrc,
                              imageSize ) );

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

    HANDLE_ERROR( hipBindTexture2D( NULL, texIn,
                                   data.dev_inSrc,
                                   desc, DIM, DIM,
                                   sizeof(float) * DIM ) );

    HANDLE_ERROR( hipBindTexture2D( NULL, texOut,
                                   data.dev_outSrc,
                                   desc, DIM, DIM,
                                   sizeof(float) * DIM ) );

    float *inputGrid = (float*)malloc( imageSize );
    for (int i=0; i<DIM*DIM; i++) {
        unputGrid[i] = 0.0f;
    }
    draw( inputGrid );

    HANDLE_ERROR( hipMemcpy( data.dev_inSrc, inputGrid,
                              imageSize,
                              hipMemcpyHostToDevice ) );
    free( inputGrid );

    bitmap.anim_and_exit( (void (*)(void*,int))anim_gpu,
                           (void (*)(void*))anim_exit );
}
